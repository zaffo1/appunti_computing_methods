
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1048576

void RandomVector(int *a, int nn){
  for (int i=0;i<nn;i++) {
    a[i]=rand()%100+1; 
  }
}

//serial sum
void VecAddSerial(int *a, int *b, int *c){
  for (int i=0;i<N;i++){
    c[i] = a[i]+b[i];
  }
}

int main(void) {
  int *h_a, *h_b, *h_c;
  int size = N*sizeof(int);

  float time;
  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  //Alloc in Host (and filling)
  h_a = (int *)malloc(size);
  h_b = (int *)malloc(size);
  h_c = (int *)malloc(size);
  RandomVector(h_a,N);
  RandomVector(h_b,N);

  //start time
  hipEventRecord(start);

  //Launch Serial Sum on CPU
  VecAddSerial(h_a,h_b,h_c);

 //stop time
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  //Print Result
  //  for(int i=0;i<N;i++){
  //  printf ("%d) h_a:%d h_b:%d h_c:%d\n",i,h_a[i],h_b[i],h_c[i]);
  //}

  //print time
  printf("Time: %3.5f ms\n",time);

  //Cleanup
  free(h_a);
  free(h_b);
  free(h_c);

  return(0);
}