#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mykernel(void) {
  printf("Hello World from GPU! (block: %d thread: %d)\n",blockIdx.x,threadIdx.x);
}

int main(void) {
  mykernel <<<3,4>>>();
  hipDeviceSynchronize();
  printf("Hello World from Host!\n");
  return 0;
}
